/*
 * 	对于寄存器的使用暂时还是属于模糊状态的
 * 	此文件用于复习学习共享内存、全局内存等
 * */



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define u32 unsigned int
#define NUM_ELEM 8

__host__ void cpu_sort(u32 * const data,
		const u32 num_elements){
	static u32 cpu_tmp_0[NUM_ELEM];
	static u32 cpu_tmp_1[NUM_ELEM];

	for(u32 bit = 0; bit<32;bit++){
		u32 base_cnt_0 = 0;
		u32 base_cnt_1 = 0;

		for(u32 i=0;i<num_elements;i++){
			const u32 d = data[i];
			const u32 bit_mask = (1<<bit);

			if((d & bit_mask) > 0){
				cpu_tmp_1[base_cnt_1]=d;
				base_cnt_1++;
			} else {
				cpu_tmp_0[base_cnt_0] = d;
				base_cnt_0++;
			}
		}
		//
		for(u32 i=0; i<base_cnt_0;i++){
			data[i] = cpu_tmp_0[i];
		}
		for(u32 i=0; i<base_cnt_1;i++){
			data[base_cnt_0+i] = cpu_tmp_1[i];
		}

	}
}


__global__ void radix_sort(u32 *const sort_tmp,
		const u32 num_lists, //1  //2
		const u32 num_elements, // 8 //4
		 u32 tid , //  0  4
		u32 * const sort_tmp_0,
		u32 * const sort_tmp_1){
	u32 idx = threadIdx.x;
	tid = idx * tid;
	printf("tid=%d\n",tid);
	for(u32 bit=0;bit<32;bit++){
		u32 base_cnt_0 = 0;
		u32 base_cnt_1 = 0;


		for(u32 i =0; i<num_elements;i++){
			const u32 elem = sort_tmp[i+tid];
			//printf("elem=%d\n",elem);
			const u32 bit_mask = (1<<bit);

			if((elem & bit_mask)>0){
				sort_tmp_1[base_cnt_1+tid]=elem;
				base_cnt_1+=num_lists;
			}else{
				sort_tmp_0[base_cnt_0+tid]=elem;
				base_cnt_0+=num_lists;
			}

			for(u32 i=0;i<base_cnt_0;i+=num_lists){
				sort_tmp[i+tid] = sort_tmp_0[i+tid];
			}
			for(u32 i=0;i<base_cnt_1;i+=num_lists){
				sort_tmp[i+base_cnt_0+tid]=sort_tmp_1[tid+i];
			}
		}
	}
	__syncthreads();
}
int main()
{
	u32 list[NUM_ELEM]={122,10,2,1,2,22,12,9};
	u32 gpu_list_cpu[8];
	clock_t start,stop;
	float costtime;
	u32 * sort_tmp_0;
	u32 * sort_tmp_1;
	u32 * gpu_list;
	hipMalloc((void**)&sort_tmp_0,8*sizeof(u32));
	hipMalloc((void**)&sort_tmp_1,8*sizeof(u32));
	hipMalloc((void**)&gpu_list,8*sizeof(u32));
	hipMemcpy(gpu_list,list,8*sizeof(u32),hipMemcpyHostToDevice);
	start = clock();
	//cpu_sort(list,8);
	radix_sort<<<1,2>>>(gpu_list,2,4,4,sort_tmp_0,sort_tmp_1);
	stop=clock();
	costtime = (float)(stop-start)/CLOCKS_PER_SEC;
	printf("花费时间为：%f\n",costtime);
	hipMemcpy(gpu_list_cpu,gpu_list,8*sizeof(u32),hipMemcpyDeviceToHost);
	for(u32 i=0;i<8;i++)
		printf("%d\n",gpu_list_cpu[i]);
	return 0;
}
