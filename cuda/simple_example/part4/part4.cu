/*
 * 常量内存
 * */
/*
 * 共享内存与同步
 * 点积运算
 * 范胜玉
 * */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define ismin(a,b) (a<b?a:b)
const int N = 4096;
const int threadPerBlock = 256;
const int blocksPerGrid = ismin(32,(N+threadPerBlock-1)/threadPerBlock);

__constant__ float dev_a[N];
__constant__  float dev_b[N];


__global__ void dot(float *c){

	__shared__ float cache[threadPerBlock];
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int cacheIndex  = threadIdx.x;
	float temp = 0;

	while(tid < N){
		temp += dev_a[tid] * dev_b[tid];
		tid += blockDim.x*gridDim.x;
	}
	cache[cacheIndex] = temp;

	//线程同步
	__syncthreads();

	//归约
	int i = blockDim.x/2;
	while(i!=0){
		if(cacheIndex<i)
			cache[cacheIndex] += cache[cacheIndex+i];
		__syncthreads();
		i/=2;
	}

	if(cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main(){
	float *a,*b,c,*partial_c;
	float * dev_partial_c;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	//cpu
	a = (float*)malloc(N*sizeof(float));
	b = (float*)malloc(N*sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	//gpu malloc
	//cudaMalloc((void**)&dev_a,N*sizeof(float));
	//cudaMalloc((void**)&dev_b,N*sizeof(float));
	hipMalloc((void**)&dev_partial_c,blocksPerGrid*sizeof(float));

	//host insert
	for(int i=0;i<N;i++){
		a[i] = i;
		b[i] = i*2;
		//printf("temp!=0 \ni = %d\n a[i]=%f\n",i,a[i]);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(dev_a),a,N*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_b),b,N*sizeof(float));
	dot<<<blocksPerGrid,threadPerBlock>>>(dev_partial_c);

	hipMemcpy(partial_c,dev_partial_c,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost);

	c = 0;
	for(int i=0;i<blocksPerGrid;i++)
		c += partial_c[i];

	float elapsedTime;
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Time to generate: %3.1f ms \n",elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#define sum_squares(x) (x*(x+1)*(2*x+1)/6)
	printf("Does GPU value %.6g = %.6g?\n",c,
			2*sum_squares((float)(N-1)));
	hipFree(dev_partial_c);

	free(a);
	free(b);
	free(partial_c);
}















