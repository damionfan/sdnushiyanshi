/*
 * 共享内存与同步
 * 点积运算
 * 范胜玉
 * */


#include <hip/hip_runtime.h>
#include <stdio.h>


#define ismin(a,b) (a<b?a:b)

const int N =  4096;
const int threadPerBlock = 256;
const int blocksPerGrid = ismin(32,(N+threadPerBlock-1)/threadPerBlock);

__global__ void dot(float *a, float *b, float *c){

	 float cache[threadPerBlock];
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	//printf("tid = %d\n a[tid]=%d\n",tid,a[tid]);
	int cacheIndex  = threadIdx.x;
	 float temp ;
	temp= 0;
	while(tid < N){
		temp += a[tid] * b[tid];
		tid += blockDim.x*gridDim.x;
	}
	cache[cacheIndex] = temp;

	//线程同步
	__syncthreads();

	//归约
	int i = blockDim.x/2;
	while(i!=0){
		if(cacheIndex<i)
			cache[cacheIndex] += cache[cacheIndex+i];
		__syncthreads();
		i/=2;
	}

	if(cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main(){
	float *a,*b,c,*partial_c;
	float *dev_a,*dev_b,*dev_partial_c;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	//cpu
	a = (float*)malloc(N*sizeof(float));
	b = (float*)malloc(N*sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	//gpu malloc
	hipMalloc((void**)&dev_a,N*sizeof(float));
	hipMalloc((void**)&dev_b,N*sizeof(float));
	hipMalloc((void**)&dev_partial_c,blocksPerGrid*sizeof(float));

	//host insert
	for(int i=0;i<N;i++){
		a[i] = i;
		b[i] = i*2;
	}

	hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice);
	dot<<<blocksPerGrid,threadPerBlock>>>(dev_a,dev_b,dev_partial_c);

	hipMemcpy(partial_c,dev_partial_c,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost);

	c = 0;
	for(int i=0;i<blocksPerGrid;i++)
		c += partial_c[i];

	float elapsedTime;
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("Time to generate: %3.1f ms \n",elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#define sum_squares(x) (x*(x+1)*(2*x+1)/6)
	printf("Does GPU value %.6g = %.6g?\n",c,
			2*sum_squares((float)(N-1)));

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);

	free(a);
	free(b);
	free(partial_c);



}












